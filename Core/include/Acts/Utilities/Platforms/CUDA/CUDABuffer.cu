#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Acts/Utilities/Platforms/CPU/CPUBuffer.hxx"

namespace Acts{

template<typename Var_t>
class CUDABuffer{

public:
  
  CUDABuffer(int size){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(Var_t));
  }

  CUDABuffer(int size, Var_t* buffer, int len, int offset=0){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(Var_t));
    CopyH2D(buffer, len, offset);
  }

  CUDABuffer(int size, const Var_t* buffer, int len, int offset=0){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(Var_t));
    CopyH2D(buffer, len, offset);
  }
  
  ~CUDABuffer(){ 
    hipFree(fDevPtr); 
  }

  Var_t* Get(int offset=0) const{ return fDevPtr+offset; }
  
  Var_t* GetHostBuffer(int len, int offset=0) const {
    Var_t* hostBuffer = new Var_t[len];
    hipMemcpy(hostBuffer, fDevPtr+offset, len*sizeof(Var_t), hipMemcpyDeviceToHost);   
    return hostBuffer;
  }

  CPUBuffer<Var_t>* GetCPUBuffer(int len, int offset=0) const {
    CPUBuffer<Var_t>* cpuBuffer = new CPUBuffer<Var_t>(len);
    hipMemcpy(cpuBuffer->Get(), fDevPtr+offset, len*sizeof(Var_t), hipMemcpyDeviceToHost);   
    return cpuBuffer;
  }
  
  // Need to test
   	Var_t& operator[](std::size_t idx)       { return fDevPtr[idx]; }
  const Var_t& operator[](std::size_t idx) const { return fDevPtr[idx]; }
  
  void CopyH2D(Var_t* buffer, int len, int offset=0){
    hipMemcpy(fDevPtr+offset, buffer, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }

  void CopyH2D(const Var_t* buffer, int len, int offset=0){
    hipMemcpy(fDevPtr+offset, buffer, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }
  
  /*
  void SetData(Var_t* buffer, int len, int offset, hipStream_t& stream){
    hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(Var_t), hipMemcpyHostToDevice,stream);
  }

  void SetData(const Var_t* buffer, int len, int offset, hipStream_t& stream){
    hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(Var_t), hipMemcpyHostToDevice,stream); 
  }
  */
  
private:
  Var_t* fDevPtr; 
  int    fSize;
};
}

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"


namespace Acts{

template<typename varT, int row, int col>
class CUDAMatrix{

public:
  
  CUDAMatrix(){ 
    //fNRows = row;
    //fNCols = col;
    hipMalloc((void**)&fDevPtr, row*col*sizeof(varT));
  }

  CUDAMatrix(varT* buffer){   
    //fNRows = row;
    //fNCols = col;
    hipMalloc((void**)&fDevPtr, row*col*sizeof(varT));
    hipMemcpy(fDevPtr, buffer, row*col*sizeof(varT), hipMemcpyHostToDevice);     
  }

  ~CUDAMatrix(){ hipFree(fDevPtr); }

  varT* dataHost() const {
    varT* hostPtr = new varT[row*col];
    hipMemcpy(hostPtr, fDevPtr, row*col*sizeof(varT), hipMemcpyDeviceToHost);   
    return hostPtr;
  }

  varT*  data() const{ return fDevPtr; }

private:

  varT* fDevPtr; 
  //int fNRows;
  //int fNCols;
  
};

}


#pragma once

#include "Acts/Utilities/Platforms/CUDA/CUDAArray.cu"
#include "Acts/Utilities/Platforms/CUDA/CPUMatrix.hpp"

namespace Acts{

template<typename Var_t>
class CUDAMatrix{

public:

  CUDAMatrix()=default;
  CUDAMatrix(size_t nRows, size_t nCols){
    fNRows = nRows;
    fNCols = nCols;
    //fDevArray = new CUDAArray<Var_t>(fNCols*fNRows);
    hipMalloc((Var_t**)&fDevPtr, fNRows*fNCols*sizeof(Var_t));
  }

  CUDAMatrix(size_t nRows, size_t nCols, CPUMatrix<Var_t>* mat){
    fNRows = nRows;
    fNCols = nCols;
    //fDevArray = new CUDAArray<Var_t>(fNCols*fNRows);
    //CopyH2D(mat->GetEl(0,0),fNRows*fNCols,0);
    hipMalloc((Var_t**)&fDevPtr, fNRows*fNCols*sizeof(Var_t));
    CopyH2D(mat->GetEl(0,0),fNRows*fNCols,0);
  }
  
  ~CUDAMatrix(){
    //delete fDevArray;
    hipFree(fDevPtr);
  }

  size_t GetNCols(){ return fNCols; }
  size_t GetNRows(){ return fNRows; }

  Var_t* GetEl(size_t row, size_t col){
    //fDevArray->Get(row+col*fNRows);
    int offset = row+col*fNRows;
    return fDevPtr+offset;
  }

  /*
  Var_t* GetHostArray(size_t len, size_t row, size_t col){
    //return fDevArray->GetHostArray(len, row+col*fNRows);
  }
  */
  
  CPUArray<Var_t>* GetCPUArray(size_t len, size_t row, size_t col){
    //return fDevArray->GetCPUArray(len, row+col*fNRows);
    int offset = row+col*fNRows;
    CPUArray<Var_t>* cpuArray = new CPUArray<Var_t>(len);
    hipMemcpy(cpuArray->Get(), fDevPtr+offset, len*sizeof(Var_t), hipMemcpyDeviceToHost);   
    return cpuArray;
  }
  
  void CopyH2D(Var_t* array, size_t len, size_t offset=0){
    //fDevArray->CopyH2D(array,len,offset);
    hipMemcpy(fDevPtr+offset, array, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }

  void CopyH2D(const Var_t* array, size_t len, size_t offset=0){
    //fDevArray->CopyH2D(array,len,offset);
    hipMemcpy(fDevPtr+offset, array, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }
  /*
  void SetColumn(size_t col, Var_t* array){
    //fDevArray->CopyH2D(array, fNRows ,col*fNRows);
  }
  */
private:
  //CUDAArray<Var_t>* fDevArray;
  Var_t* fDevPtr; 
  size_t fNCols;
  size_t fNRows;
};

}


#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

namespace Acts{

template<typename varT>
class CUDABuffer{

public:
  
  CUDABuffer(int size){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(varT));
  }

  SetData(varT* buffer, int len, int offset=0, hipStream_t stream=NULL){
    if (stream != NULL){
      hipStreamCreate(&stream);
      hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice,stream); 
    }
    else if (stream == NULL) {
	hipMemcpy(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice);
    }
  }

  ~CUDABuffer(){ 
    hipFree(fDevPtr); 
  }

  varT* dataHost(int len, int offset=0) const {
    varT* hostPtr = new varT[len];
    hipMemcpy(hostPtr, fDevPtr+offset, len*sizeof(varT), hipMemcpyDeviceToHost);   
    return hostPtr;
  }

  varT* data(int offset=0) const{ return fDevPtr+offset; }

private:

  varT* fDevPtr; 
  int   fSize;
};
}

/*
template<typename varT, int row, int col>
class CUDAMatrix{

public:
  
  CUDAMatrix(){ 
    fNRows = row;
    fNCols = col;
    hipMalloc((void**)&fDevPtr, row*col*sizeof(varT));
    hipStreamCreate(&fStream);
  }

  CUDAMatrix(varT* buffer){   
    fNRows = row;
    fNCols = col;
    hipMalloc((void**)&fDevPtr, row*col*sizeof(varT));
    hipStreamCreate(&fStream);
    hipMemcpyAsync(fDevPtr, buffer, row*col*sizeof(varT), hipMemcpyHostToDevice,fStream); 
    
    std::cout << "Create Matrix" << std::endl;
  }

  ~CUDAMatrix(){ 
    hipFree(fDevPtr); 
    hipStreamDestroy(fStream);
    std::cout << "Destroy Matrix" << std::endl;
  }

  varT* dataHost() const {
    varT* hostPtr = new varT[row*col];
    hipMemcpy(hostPtr, fDevPtr, row*col*sizeof(varT), hipMemcpyDeviceToHost);   
    return hostPtr;
  }

  varT*  data() const{ return fDevPtr; }

private:

  varT* fDevPtr; 
  int fNRows;
  int fNCols;
  hipStream_t fStream;
*/


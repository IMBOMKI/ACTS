#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

namespace Acts{

template<typename varT>
class CUDABuffer{

public:
  
  CUDABuffer(int size){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(varT));
  }

  SetData(varT* buffer, int len, int offset, hipStream_t& stream){
    hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice,stream);
  }

  SetData(const varT* buffer, int len, int offset, hipStream_t& stream){
    hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice,stream); 
  }

  SetData(varT* buffer, int len, int offset=0){
    hipMemcpy(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice);
  }

  SetData(const varT* buffer, int len, int offset=0){
    hipMemcpy(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice);
  }

  ~CUDABuffer(){ 
    hipFree(fDevPtr); 
  }

  varT* dataHost(int len, int offset=0) const {
    varT* hostPtr = new varT[len];
    hipMemcpy(hostPtr, fDevPtr+offset, len*sizeof(varT), hipMemcpyDeviceToHost);   
    return hostPtr;
  }

  varT* data(int offset=0) const{ return fDevPtr+offset; }

private:

  varT* fDevPtr; 
  int   fSize;
};
}

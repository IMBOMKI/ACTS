#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"


namespace Acts{

template<typename Var_t>
class CUDABuffer{

public:
  
  CUDABuffer(int size){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(Var_t));
  }

  CUDABuffer(int size, Var_t* buffer, int offset=0){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(Var_t));
    hipMemcpy(fDevPtr+offset, buffer, fSize*sizeof(Var_t), hipMemcpyHostToDevice);
  }
  
  CUDABuffer(int size, const Var_t* buffer, int offset=0){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(Var_t));
    hipMemcpy(fDevPtr+offset, buffer, fSize*sizeof(Var_t), hipMemcpyHostToDevice);
  }

  void SetData(Var_t* buffer, int len, int offset=0){
    hipMemcpy(fDevPtr+offset, buffer, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }

  void SetData(const Var_t* buffer, int len, int offset=0){
    hipMemcpy(fDevPtr+offset, buffer, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }
  
  void SetData(Var_t* buffer, int len, int offset, hipStream_t& stream){
    hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(Var_t), hipMemcpyHostToDevice,stream);
  }

  void SetData(const Var_t* buffer, int len, int offset, hipStream_t& stream){
    hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(Var_t), hipMemcpyHostToDevice,stream); 
  }

  ~CUDABuffer(){ 
    hipFree(fDevPtr); 
  }

  Var_t* dataHost(int len, int offset=0) const {
    Var_t* hostPtr = new Var_t[len];
    hipMemcpy(hostPtr, fDevPtr+offset, len*sizeof(Var_t), hipMemcpyDeviceToHost);   
    return hostPtr;
  }
  
  Var_t* data(int offset=0) const{ return fDevPtr+offset; }

private:

  Var_t* fDevPtr; 
  int   fSize;
};
}

#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"


namespace Acts{

template<typename varT>
class CUDABuffer{

public:
  
  CUDABuffer(int size){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(varT));
  }

  CUDABuffer(int size, varT* buffer, int offset=0){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(varT));
    hipMemcpy(fDevPtr+offset, buffer, fSize*sizeof(varT), hipMemcpyHostToDevice);
  }
  
  CUDABuffer(int size, const varT* buffer, int offset=0){ 
    fSize = size;
    hipMalloc((void**)&fDevPtr, fSize*sizeof(varT));
    hipMemcpy(fDevPtr+offset, buffer, fSize*sizeof(varT), hipMemcpyHostToDevice);
  }
  
  void SetData(varT* buffer, int len, int offset, hipStream_t& stream){
    hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice,stream);
  }

  void SetData(const varT* buffer, int len, int offset, hipStream_t& stream){
    hipMemcpyAsync(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice,stream); 
  }

  void SetData(varT* buffer, int len, int offset=0){
    hipMemcpy(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice);
  }

  void SetData(const varT* buffer, int len, int offset=0){
    hipMemcpy(fDevPtr+offset, buffer, len*sizeof(varT), hipMemcpyHostToDevice);
  }

  ~CUDABuffer(){ 
    hipFree(fDevPtr); 
  }

  varT* dataHost(int len, int offset=0) const {
    varT* hostPtr = new varT[len];
    hipMemcpy(hostPtr, fDevPtr+offset, len*sizeof(varT), hipMemcpyDeviceToHost);   
    return hostPtr;
  }

  varT* data(int offset=0) const{ return fDevPtr+offset; }

private:

  varT* fDevPtr; 
  int   fSize;
};
}
